#include "hip/hip_runtime.h"
//
// Created by Jasmine Lu on 2019-04-27.
//

#include <string>
#include <iostream>
#include <sstream>
#include <iomanip>
#include "GPiNeuron.h"
#include "gating.h"


__device__ 
void compute_next_state(gpi_state_t *in, gpi_state_t *out, gpi_param_t *params, double dt){
    compute_currents(in, out, params);
    compute_gating(in, out, params, dt);
    
    out -> voltage = params -> g_L;
    double current_sum =
            out -> I_L +
            out -> I_Na +
            out -> I_K +
            out -> I_T +
            out -> I_Ca +
            out -> I_ahp;
    out -> voltage = in -> voltage + dt * current_sum / params->C_m;
}
__device__ 
void compute_currents(gpi_state_t *in, gpi_state_t *out, gpi_param_t *params){
    double v = in -> voltage;
    out->I_L = -(params->g_L * (v - params->E_L));
    out->I_Na = -(params->g_Na * (pow(gpe_minf(v), 3)) * in->H * (v - params->E_Na));
    out->I_K = -(params->g_K * pow(in->N, 4) * (v - params->E_K));
    out->I_T = -(params->g_T * pow(gpe_ainf(v), 3) * in->R * (v - params->E_Ca));
    out->I_Ca = -(params->g_Ca * pow(gpe_sinf(v), 3) * (v - params->E_Ca));
    out->I_ahp = -(params->g_ahp * (v - params->E_ahp) * (in->CA/(in->CA + 10)));
}


__device__ 
void compute_gating(gpi_state_t *in, gpi_state_t *out, gpi_param_t *params, double dt){
    double v = in->voltage;
    out->H= in->H + dt * 0.05 * ((gpe_hinf(v) - in->H)/gpe_tauh(v));
    out->R= in->R + dt * 1.0 * (gpe_rinf(v) - in->R)/30;
    out->N= in->N+ dt * 0.1 * (gpe_ninf(v) - in->N)/gpe_taun(v);
    out->CA = in->CA + dt * pow(10, -4) * (-(in->I_Ca) - in->I_T - (15 * in->CA));
}


void init_state(gpi_state_t *in){
    in->voltage = -57.0;
    double v = in->voltage;
    in->H = gpe_hinf(v);
    in->R = gpe_rinf(v);
    in->N = gpe_ninf(v);
    in->CA = 0.1;
}


void init_gpi_param(gpi_param_t *param){
    param->C_m = 1.0;
    param->g_L = 0.1;
    param->E_L = -65.0;
    param->g_Na = 120.0;
    param->E_Na = 55.0;
    param->g_K = 30.0;
    param->E_K = -80.0;
    param->g_T = 0.5;
    param->E_T = 0.0;
    param->g_Ca = 0.15;
    param->E_Ca = 120.0;
    param->g_ahp = 10.0;
    param->E_ahp = -80.0;
}


std::string get_debug_string(gpi_state_t* state) {
    std::ostringstream debug_str;
    debug_str.setf(std::ios::fixed, std::ios::floatfield);
    debug_str << std::setprecision(15);
    debug_str
        << "VOLTAGE=" << state->voltage
        << ", I_L=" << state->I_Na
        << ", I_Na= " << state->I_L
        << ", I_K=" << state->I_K
        << ", I_T=" << state->I_T
        << ", I_Ca=" << state->I_Ca
        << ", I_ahp=" << state->I_ahp
        << ", H=" << state->H
        << ", R=" << state->R
        << ", N=" << state->N
        << ", CA=" << state->CA;
    return debug_str.str();
}
