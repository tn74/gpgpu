#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "gating.h"

__device__ double gpe_ainf(double V)
{
  return 1/(1+exp(-(V+57)/2));
}

__device__ double gpe_hinf(double V)
{
  return 1/(1+exp((V+58)/12));
}

__device__ double gpe_minf(double V)
{
  return 1/(1+exp(-(V+37)/10));
}

__device__ double gpe_ninf(double V)
{
  return 1/(1+exp(-(V+50)/14));
}

__device__ double gpe_rinf(double V)
{
  return 1/(1+exp((V+70)/2));
}

__device__ double gpe_sinf(double V)
{
  return 1/(1+exp(-(V+35)/2));
}

__device__ double gpe_tauh(double V)
{
  return 0.05+0.27/(1+exp(-(V+40)/-12));
}

__device__ double gpe_taun(double V)
{
  return 0.05+0.27/(1+exp(-(V+40)/-12));
}

__device__ double Hinf(double V)
{
  return 1/(1+exp(-(V+57)/2));
}

__device__ double stn_ainf(double V)
{
  return 1/(1+exp(-(V+63)/7.8));
}

__device__ double stn_binf(double R)
{
  return 1/(1+exp(-(R-0.4)/0.1))-1/(1+exp(0.4/0.1));
}

__device__ double stn_cinf(double V)
{
  return 1/(1+exp(-(V+20)/8));
}

__device__ double stn_hinf(double V)
{
  return 1/(1+exp((V+39)/3.1));
}

__device__ double stn_minf(double V)
{
  return 1/(1+exp(-(V+30)/15));
}

__device__ double stn_ninf(double V)
{
  return 1/(1+exp(-(V+32)/8.0));
}

__device__ double stn_rinf(double V)
{
  return 1/(1+exp((V+67)/2));
}

__device__ double stn_sinf(double V)
{
  return 1/(1+exp(-(V+39)/8));
}

__device__ double stn_tauc(double V)
{
  return 1 + 10/(1+exp(-(V+80)/26));
}

__device__ double stn_tauh(double V)
{
  return 1 + 500/(1+exp(-(V+57)/-3));
}

__device__ double stn_taun(double V)
{
  return 1 + 100/(1+exp(-(V+80)/-26));
}

__device__ double stn_taur(double V)
{
  return 7.1 + 17.5/(1+exp(-(V-68)/-2.2));
}

__device__ double th_hinf(double V)
{
  return 1/(1+exp((V+41)/4));
}

__device__ double th_minf(double V)
{
  return 1/(1+exp(-(V+37)/7));
}

__device__ double th_pinf(double V)
{
  return 1/(1+exp(-(V+60)/6.2));
}

__device__ double th_rinf(double V)
{
  return 1/(1+exp((V+84)/4));
}

__device__ double ah(double V)
{
  return 0.128*exp(-(V+46)/18);
}

__device__ double bh(double V)
{
  return 4/(1+exp(-(V+23)/5));
}

__device__ double th_tauh(double V)
{
  return 1/(ah(V)+bh(V));
}

__device__ double th_taur(double V)
{
  return 0.15*(28+exp(-(V+25)/10.5));
}
