#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "gating.h"


#ifdef __HIPCC__
#define CUDA_HOSTDEV __host__ __device__
#else
#define CUDA_HOSTDEV
#endif

CUDA_HOSTDEV double gpe_ainf(double V)
{
  return 1/(1+exp(-(V+57)/2));
}

CUDA_HOSTDEV double gpe_hinf(double V)
{
  return 1/(1+exp((V+58)/12));
}

CUDA_HOSTDEV double gpe_minf(double V)
{
  return 1/(1+exp(-(V+37)/10));
}

CUDA_HOSTDEV double gpe_ninf(double V)
{
  return 1/(1+exp(-(V+50)/14));
}

CUDA_HOSTDEV double gpe_rinf(double V)
{
  return 1/(1+exp((V+70)/2));
}

CUDA_HOSTDEV double gpe_sinf(double V)
{
  return 1/(1+exp(-(V+35)/2));
}

CUDA_HOSTDEV double gpe_tauh(double V)
{
  return 0.05+0.27/(1+exp(-(V+40)/-12));
}

CUDA_HOSTDEV double gpe_taun(double V)
{
  return 0.05+0.27/(1+exp(-(V+40)/-12));
}

CUDA_HOSTDEV double Hinf(double V)
{
  return 1/(1+exp(-(V+57)/2));
}

CUDA_HOSTDEV double stn_ainf(double V)
{
  return 1/(1+exp(-(V+63)/7.8));
}

CUDA_HOSTDEV double stn_binf(double R)
{
  return 1/(1+exp(-(R-0.4)/0.1))-1/(1+exp(0.4/0.1));
}

CUDA_HOSTDEV double stn_cinf(double V)
{
  return 1/(1+exp(-(V+20)/8));
}

CUDA_HOSTDEV double stn_hinf(double V)
{
  return 1/(1+exp((V+39)/3.1));
}

CUDA_HOSTDEV double stn_minf(double V)
{
  return 1/(1+exp(-(V+30)/15));
}

CUDA_HOSTDEV double stn_ninf(double V)
{
  return 1/(1+exp(-(V+32)/8.0));
}

CUDA_HOSTDEV double stn_rinf(double V)
{
  return 1/(1+exp((V+67)/2));
}

CUDA_HOSTDEV double stn_sinf(double V)
{
  return 1/(1+exp(-(V+39)/8));
}

CUDA_HOSTDEV double stn_tauc(double V)
{
  return 1 + 10/(1+exp(-(V+80)/26));
}

CUDA_HOSTDEV double stn_tauh(double V)
{
  return 1 + 500/(1+exp(-(V+57)/-3));
}

CUDA_HOSTDEV double stn_taun(double V)
{
  return 1 + 100/(1+exp(-(V+80)/-26));
}

CUDA_HOSTDEV double stn_taur(double V)
{
  return 7.1 + 17.5/(1+exp(-(V-68)/-2.2));
}

CUDA_HOSTDEV double th_hinf(double V)
{
  return 1/(1+exp((V+41)/4));
}

CUDA_HOSTDEV double th_minf(double V)
{
  return 1/(1+exp(-(V+37)/7));
}

CUDA_HOSTDEV double th_pinf(double V)
{
  return 1/(1+exp(-(V+60)/6.2));
}

CUDA_HOSTDEV double th_rinf(double V)
{
  return 1/(1+exp((V+84)/4));
}

CUDA_HOSTDEV double ah(double V)
{
  return 0.128*exp(-(V+46)/18);
}

CUDA_HOSTDEV double bh(double V)
{
  return 4/(1+exp(-(V+23)/5));
}

CUDA_HOSTDEV double th_tauh(double V)
{
  return 1/(ah(V)+bh(V));
}

CUDA_HOSTDEV double th_taur(double V)
{
  return 0.15*(28+exp(-(V+25)/10.5));
}
