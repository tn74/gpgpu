#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-04-04.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>

#include "BGNetwork.h"
#include "THNeuron.h"

__global__ 
void advance_step(void** start_state, void** end_state, void** params, int* cell_counts, double dt) {
    // Determine what start to end this is supposed to compute
    int cell_ind = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;
    int cell_type = blockIdx.x;
    if (cell_ind >= cell_counts[cell_type]) {return;}

    // Execute based on cell type
    if (cell_type == TH) {
        th_state_t start = (th_state_t) start_state[TH][cell_ind];
        auto end = (th_state_t) end_state[TH][cell_ind];
        auto param = (th_params_t) params[TH];
    }

    compute_next_state(&start, &end, param, dt);
}

BGNetwork::BGNetwork(simulation_parameters_t* sp){
    std::cout << "BGNetwork Constructor" << std::endl;
    sim_params = sp;
    dt_index = 0;
    
    params =        (void**) hipMalloc(CELL_TYPE_COUNT * sizeof(void*));
    start_st =      (void**) hipMalloc(CELL_TYPE_COUNT * sizeof(void*));
    end_st =        (void**) hipMalloc(CELL_TYPE_COUNT * sizeof(void*)); 
    cell_counts =   (int*) cudaMallox(CELL_TYPE_COUNT * sizeof(int*));
    for (int c = 0; c < CELL_TYPE_COUNT; ++c) {cell_counts[c] = sp->cells_per_type;}

    state_start[TH] = (hipMalloc(sp->cells_per_type * sizeof(th_state)));
    state_end[TH] = (hipMalloc(sp->cells_per_type * sizeof(th_state)));
    
    build_parameter_map();
    initialize_cells();

}

void BGNetwork::build_parameter_map() {
    params[TH] = hipMalloc(sizeof(th_param));
    th_params = (th_param_t *) params[TH];
    th_params -> C_m = 1.0;
    th_params -> g_L = 0.05;
    th_params -> E_L = -70;
    th_params -> g_Na = 3.0;
    th_params -> E_Na = 50.0;
    th_params -> g_K = 5.0;
    th_params -> E_K = -75.0;
    th_params -> g_T = 5.0;
    th_params -> E_T = 0.0;
    std::cout << "Built Parameter Map" << std::endl;
}

void BGNetwork::initialize_cells() {
    for (int i = 0; i < sim_params-> cells_per_type; ++i) {
        init_state(&state_start[i]);
    }
}

void BGNetwork::advance_time_step() {
    int* 
    for (int i = 0; i < (sim_params-> cells_per_type); ++i) {
        compute_next_state<<<1, 16>>>(&state_start[i], &state_end[i], th_params, sim_params->dt);
    }
    dt_index ++;
    th_state_t* tmp = state_start;
    state_start = state_end;
    state_end = tmp;
    std::cout << "\r"  << "Iteration: " << dt_index << std::endl;
}

void BGNetwork::debug(th_state_t* state) {
    for (int i = 0; i < sim_params->cells_per_type; ++i) {
        std::ostringstream output_file_name;
        output_file_name << "output/TH_NEURON_" << i << ".txt";
        std::ofstream out(output_file_name.str(), std::ios::app);
        out << "DT=" << dt_index << ", " << get_debug_string(&state[i]) << std::endl;
        out.close();
    }
}
int BGNetwork::simulate() {
    system("exec rm -r output/*");
    debug(state_start);
    for (int i = 0; i < sim_params->duration / sim_params->dt; ++i) {
        advance_time_step();
        debug(state_start);
    }
    return 0;
};
