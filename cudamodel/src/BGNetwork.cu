#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-04-04.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>

#include "BGNetwork.h"
#include "THNeuron.h"

__global__ 
 void advance_step(void** start_state, void** end_state, void** params, int* cell_counts, double dt, int tbp) {   
    // Determine what start to end this is supposed to compute
    int cell_ind = blockIdx.y * tbp + threadIdx.x;
    int cell_type = blockIdx.x;
    if (cell_ind >= cell_counts[cell_type]) {return;}

    // Execute based on cell type
//    if (cell_type == TH) {
        auto start = ((th_state_t**)start_state)[TH][cell_ind];
        auto end = ((th_state_t**) end_state)[TH][cell_ind];
        auto param = ((th_param_t**) params)[TH];
//    }

    compute_next_state(&start, &end, param, dt);
}


BGNetwork::BGNetwork(simulation_parameters_t* sp){
    std::cout << "BGNetwork Constructor" << std::endl;
    sim_params = sp;
    dt_index = 0;
    // params =        (void**) hipMalloc(CELL_TYPE_COUNT * sizeof(void*));i
    size_t CTC_SIZE = CELL_TYPE_COUNT * sizeof(void*);
    params = (void**) malloc(CTC_SIZE); hipMallocManaged(&params, CTC_SIZE);
    start_st = (void**) malloc(CTC_SIZE); hipMallocManaged(&end_st, CTC_SIZE);
    end_st  = (void**) malloc(CTC_SIZE); hipMallocManaged(&start_st, CTC_SIZE);
    std::cout << " OG " << cell_counts << std::endl;
    cell_counts  = (int*) malloc(CELL_TYPE_COUNT * sizeof(int*)); hipMallocManaged(&cell_counts, CELL_TYPE_COUNT * sizeof(int*));
    std::cout << cell_counts << " " << cell_counts + 1 <<  std::endl;
    std::cout << *cell_counts << std::endl;
    for (int c = 0; c < CELL_TYPE_COUNT; ++c) {cell_counts[c] = sp->cells_per_type;}

    start_st[TH] = malloc(cell_counts[TH] * sizeof(th_state));
    end_st[TH] = malloc(cell_counts[TH]  * sizeof(th_state));
    
    build_parameter_map();
    initialize_cells();

}

void BGNetwork::build_parameter_map() {
    params[TH] = malloc(sizeof(th_param));
    auto th_params = (th_param_t *) params[TH];
    th_params -> C_m = 1.0;
    th_params -> g_L = 0.05;
    th_params -> E_L = -70;
    th_params -> g_Na = 3.0;
    th_params -> E_Na = 50.0;
    th_params -> g_K = 5.0;
    th_params -> E_K = -75.0;
    th_params -> g_T = 5.0;
    th_params -> E_T = 0.0;
    std::cout << "Built Parameter Map" << std::endl;
}

void BGNetwork::initialize_cells() {
    auto th_start = (th_state_t*) start_st[TH];
    for (int i = 0; i < sim_params-> cells_per_type; ++i) {
        init_state(&th_start[i]);
    }
}

void BGNetwork::advance_time_step() {
    dim3 grid(1, sim_params -> cells_per_type / 64 + 1);
    advance_step<<<grid, 64>>>(start_st, end_st, params, cell_counts, sim_params->dt, THREADS_PER_BLOCK);
    hipDeviceSynchronize();
    dt_index ++;
    void** tmp = start_st;
    start_st = end_st;
    end_st = tmp;
    std::cout << "\r"  << "Iteration: " << dt_index << std::endl;
}

void BGNetwork::debug(th_state_t* state) {
    for (int i = 0; i < sim_params->cells_per_type; ++i) {
        std::ostringstream output_file_name;
        output_file_name << "output/TH_NEURON_" << i << ".txt";
        std::ofstream out(output_file_name.str(), std::ios::app);
        out << "DT=" << dt_index << ", " << get_debug_string(&state[i]) << std::endl;
        out.close();
    }
}
int BGNetwork::simulate() {
    system("exec rm -r output/*");
    //debug(state_start);
    for (int i = 0; i < sim_params->duration / sim_params->dt; ++i) {
        advance_time_step();
    //    debug(state_start);
    }
    return 0;
};
