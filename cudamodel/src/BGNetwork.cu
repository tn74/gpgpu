#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-04-04.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include "BGNetwork.h"
#include "THNeuron.h"
#include <chrono>
#include <thread>
__global__ 
 void advance_step(void** start_state, void** end_state, void** params, int* cell_counts, double dt, int tbp) {   
     // Determine what start to end this is supposed to compute
    int cell_ind = blockIdx.y * tbp + threadIdx.x;
    int cell_type = blockIdx.x;
    if (cell_ind >= cell_counts[cell_type]) {return;}
    
    // Execute based on cell type
//    if (cell_type == TH) {
        th_state_t* start = &((th_state_t**)start_state)[TH][cell_ind];
        th_state_t* end = &((th_state_t**) end_state)[TH][cell_ind];
        th_param_t* param = ((th_param_t**) params)[TH];
//    }
    compute_next_state(start, end, param, dt);
}


void BGNetwork::init_states() {
    int cell_counts = this -> sim_params -> cells_per_type;
    this -> states = (void***) malloc(STATE_COUNT * sizeof(void*));
    for (int i = 0; i < STATE_COUNT; ++i) {
        this->states[i] = (void**) malloc(CELL_TYPE_COUNT * sizeof(void*));
        this->states[i][TH] = (void*) malloc(cell_counts * sizeof(th_state_t*));
        hipMallocManaged(&(this->states[i]), CELL_TYPE_COUNT * sizeof(void*));
        hipMallocManaged(&(this->states[i][TH]), cell_counts * sizeof(th_state_t*));
    }
}

void BGNetwork::init_parameters() {
    this->params = (void**) malloc(CELL_TYPE_COUNT * sizeof(void*)); 
    hipMallocManaged(&(this->params), CELL_TYPE_COUNT * sizeof(void*)); 
    this -> init_th_param();
}

void BGNetwork::init_result_structures() {
    int cell_counts = this->sim_params->cells_per_type;
    double total_dt = this->sim_params->duration / this->sim_params->dt; 
    
    this -> voltage = (double***) malloc(sizeof(double**));
    for (int k = 0; k < CELL_TYPE_COUNT; ++k) { 
        (this->voltage)[k]       = (double**) malloc(cell_counts * sizeof(double*));
        (this->debug_states)[k]  = (void**) malloc(cell_counts * sizeof(void*));
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            (this->voltage)[k][cell_ind] = (double*) malloc(total_dt * sizeof(double));
            if (k == TH){(this->debug_states)[k][cell_ind] = (th_state_t*) malloc(total_dt * sizeof(th_state_t));}
        }
    }
}

BGNetwork::BGNetwork(simulation_parameters_t* sp){
    std::cout << "BGNetwork Constructor" << std::endl;
    this -> sim_params = sp;
    this -> dt = 0;
    this -> init_states();
    this -> init_parameters();     
    this -> initialize_cells();

}

void BGNetwork::init_th_param() {
    this->params[TH] = malloc(sizeof(th_param));
    hipMallocManaged(&params[TH], sizeof(th_param));
    auto th_params = (th_param_t *) params[TH];
    th_params -> C_m = 1.0;
    th_params -> g_L = 0.05;
    th_params -> E_L = -70;
    th_params -> g_Na = 3.0;
    th_params -> E_Na = 50.0;
    th_params -> g_K = 5.0;
    th_params -> E_K = -75.0;
    th_params -> g_T = 5.0;
    th_params -> E_T = 0.0;
    std::cout << "Built Parameter Map" << std::endl;
}

void BGNetwork::initialize_cells() {
    auto th_start = (th_state_t*) start_st[TH];
    for (int i = 0; i < sim_params-> cells_per_type; ++i) {
        init_state(&th_start[i]);
    }
}

void BGNetwork::advance_time_step() {
    dim3 grid(CELL_TYPE_COUNT, sim_params -> cells_per_type / THREADS_PER_BLOCK + 1);
    advance_step<<<grid, THREADS_PER_BLOCK>>>(start_st, end_st, params, cell_counts, sim_params->dt, THREADS_PER_BLOCK);
    hipDeviceSynchronize();
    //std::cout << "Out of CUDA Call" << std::endl; 
    //std::cout << ((th_state_t**) start_st) << ", " << ((th_state_t**) start_st)[0][0].voltage << std::endl; 
    //std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    for (int cell_type = 0; cell_type < CELL_TYPE_COUNT; ++cell_type) {
        for (int cell_ind = 0; cell_ind < sim_params->cells_per_type; ++cell_ind) { // Strangely cell_counts[TH] does not work here?!?! 
            VOLTAGE[cell_type][cell_ind][dt_index] = ((th_state_t**)start_st)[cell_type][cell_ind].voltage;
        }
    }
    dt_index ++;
    void** tmp = start_st;
    start_st = end_st;
    end_st = tmp;
    //std::cout << "\r"  << "Iteration: " << dt_index << std::endl;
}

void BGNetwork::debug(th_state_t* state) {
    for (int i = 0; i < sim_params->cells_per_type; ++i) {
        std::ostringstream output_file_name;
        output_file_name << "output/TH_NEURON_" << i << ".txt";
        std::ofstream out(output_file_name.str(), std::ios::app);
        out << "DT=" << dt_index << ", " << get_debug_string(&state[i]) << std::endl;
        out.close();
    }
}

int BGNetwork::simulate() {
    for (int i = 0; i < sim_params->duration / sim_params->dt; ++i) {
        advance_time_step();
    }
    std::cout<< "End of Simulate" << std::endl;
    return 0;
};

double*** BGNetwork::get_voltage() {
    return VOLTAGE;
}
