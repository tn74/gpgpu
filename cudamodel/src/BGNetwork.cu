#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-04-04.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include "BGNetwork.h"
#include "THNeuron.h"
#include <chrono>
#include <thread>
__global__ 
 void advance_step(void*** states, void** params, int number_of_cells,  double dt, int tbp, int timesteps_to_run) {   
    int ctc = 4;
    int cell_ind = (blockIdx.x * tbp + threadIdx.x) / ctc;
    int cell_type = (blockIdx.x * tbp + threadIdx.x) % ctc;
    //(((th_state_t***)states)[0][0] + 1) -> voltage = 1; 
    // Execute based on cell type
    if (cell_ind >= number_of_cells){return;}
    if (cell_type == 0) {
        th_state_t* state_series = ((th_state_t***)states)[cell_type][cell_ind];
        //(state_series + 1) -> voltage = 2;
        th_param_t* param = (th_param_t*) params[0];
        //(state_series + 1) -> voltage = 3;
        for (int timestep = 0; timestep < timesteps_to_run; ++timestep) {
            //(&state_series[0]) -> voltage = 100 + timesteps_to_run;
            compute_next_state(state_series + timestep, state_series + timestep + 1, param, dt);
        }
    }
 }


void BGNetwork::init_states() {
    int cell_counts = this -> sim_params -> cells_per_type;
    this -> states = (void****) malloc(STATE_COUNT * sizeof(void***));
    for (int i = 0; i < STATE_COUNT; ++i) {
        this->states[i] = (void***) malloc(CELL_TYPE_COUNT * sizeof(void***));
        this->states[i][TH] = (void**) malloc(cell_counts * sizeof(void**));
        hipMallocManaged(&(this->states[i]), CELL_TYPE_COUNT * sizeof(void**));
        hipMallocManaged(&(this->states[i][TH]), cell_counts * sizeof(void*));
        std::cout << "Assigned and Cuda Malloc Managed STATE " << i << " TH \n";
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            std::cout << "Size of th_state_t " << sizeof(th_state_t) << " bytes" << "Size of th_state " << sizeof(th_state) << " bytes\n";    
            this->states[i][TH][cell_ind] = (th_state_t*) malloc(STEPS_PER_THREAD * sizeof(th_state));
            hipMallocManaged(&(this->states[i][TH][cell_ind]), STEPS_PER_THREAD * sizeof(th_state));
        }
    }
}

void BGNetwork::init_parameters() {
    this->params = (void**) malloc(CELL_TYPE_COUNT * sizeof(void*)); 
    hipMallocManaged(&(this->params), CELL_TYPE_COUNT * sizeof(void*)); 
    this -> init_th_param();
}

void BGNetwork::init_result_structures() {
    int cell_counts = this->sim_params->cells_per_type;
    double total_dt = this->sim_params->duration / this->sim_params->dt; 
    
    this -> voltage = (double***) malloc(sizeof(double**));
    this -> debug_states = (void***) malloc(sizeof(double**));
    for (int k = 0; k < CELL_TYPE_COUNT; ++k) { 
        (this->voltage)[k]       = (double**) malloc(cell_counts * sizeof(double*));
        (this->debug_states)[k]  = (void**) malloc(cell_counts * sizeof(void*));
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            (this->voltage)[k][cell_ind] = (double*) malloc(total_dt * sizeof(double));
            if (k == TH){(this->debug_states)[k][cell_ind] = (th_state_t*) malloc(total_dt * sizeof(th_state_t));}
        }
    }
}

BGNetwork::BGNetwork(simulation_parameters_t* sp){
    std::cout << "BGNetwork Constructor" << std::endl;
    this -> sim_params = sp;
    this -> dt = 0;
    this -> init_states();
    std::cout << "Finished Init States" << std::endl;
    this -> init_parameters();     
    std::cout << "Initialized Parameters" << std::endl;
    this -> initialize_cells();
    std::cout << "Initialized Cells" << std::endl;
    this -> init_result_structures();
    std::cout << "Initialized Result Structures" << std::endl;
}

void BGNetwork::init_th_param() {
    this->params[TH] = malloc(sizeof(th_param_t));
    hipMallocManaged(&params[TH], sizeof(th_param_t));
    auto th_params = (th_param_t *) params[TH];
    th_params -> C_m = 1.0;
    th_params -> g_L = 0.05;
    th_params -> E_L = -70;
    th_params -> g_Na = 3.0;
    th_params -> E_Na = 50.0;
    th_params -> g_K = 5.0;
    th_params -> E_K = -75.0;
    th_params -> g_T = 5.0;
    th_params -> E_T = 0.0;
    std::cout << "Built Parameter Map" << std::endl;
}

void BGNetwork::initialize_cells() {
    auto th_start = (th_state_t**) this->states[0][TH];
    for (int i = 0; i < sim_params-> cells_per_type; ++i) {
        init_state(&th_start[i][0]); //0'th time index of each cell
    }
}

void BGNetwork::advance_time_step() {
    int executions = this->dt / STEPS_PER_THREAD;
    void*** sim_state = this->states[executions  % STATE_COUNT];
    void*** rest_state = this->states[(executions + 1) % STATE_COUNT];
    int block_count = (sim_params->cells_per_type * CELL_TYPE_COUNT) / THREADS_PER_BLOCK;
    if ((sim_params->cells_per_type * CELL_TYPE_COUNT) % THREADS_PER_BLOCK > 0){block_count ++;}
    
    std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
    advance_step<<<block_count, THREADS_PER_BLOCK>>>(sim_state, params, sim_params->cells_per_type, sim_params->dt, THREADS_PER_BLOCK, STEPS_PER_THREAD - 1);
    hipDeviceSynchronize();
    std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
    /*
    for (int type = 0; type < CELL_TYPE_COUNT; ++type){
        for (int cell_ind = 0; this->sim_params->cells_per_type; ++cell_ind){ 
            rest_state[type][cell_ind][0] = ((th_state_t*) sim_state)[type][cell_ind][STEPS_PER_THREAD - 1];
        }
    }
    */
    dt += STEPS_PER_THREAD;
    //std::cout << "Out of CUDA Call" << std::endl; 
    //std::cout << ((th_state_t**) start_st) << ", " << ((th_state_t**) start_st)[0][0].voltage << std::endl; 
    //std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    /*
    for (int cell_type = 0; cell_type < CELL_TYPE_COUNT; ++cell_type) {
        for (int cell_ind = 0; cell_ind < sim_params->cells_per_type; ++cell_ind) { // Strangely cell_counts[TH] does not work here?!?! 
            VOLTAGE[cell_type][cell_ind][dt_index] = ((th_state_t**)start_st)[cell_type][cell_ind].voltage;
        }
    }
    dt_index ++;
    void** tmp = start_st;
    start_st = end_st;
    end_st = tmp;*/
    //std::cout << "\r"  << "Iteration: " << dt_index << std::endl;
}
/*
int BGNetwork::debug(th_state_t* state) {
    for (int i = 0; i < sim_params->cells_per_type; ++i) {
        std::ostringstream output_file_name;
        output_file_name << "output/TH_NEURON_" << i << ".txt";
        std::ofstream out(output_file_name.str(), std::ios::app);
        out << "DT=" << dt << ", " << get_debug_string(&state[i]) << std::endl;
        out.close();
    }
    return 0;
}
*/
int BGNetwork::simulate() {
    for (int i = 0; i < sim_params->duration / sim_params->dt; ++i) {
        advance_time_step();
    }
    std::cout<< "End of Simulate" << std::endl;
    return 0;
}


int BGNetwork::simulate_debug() {
    for (int i = 0; i < sim_params->duration / sim_params->dt; ++i) {
        advance_time_step();
    }
    std::cout<< "End of Simulate Debug" << std::endl;
    return 0;
}


double*** BGNetwork::get_voltage() {
    return this->voltage;
}


void*** BGNetwork::get_debug_states() {
    return this -> debug_states;
}
