#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-04-04.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include "BGNetwork.h"
#include "THNeuron.h"
#include "STNNeuron.h"

__global__ 
 void advance_step(void*** states, void** params, int number_of_cells,  double dt, int tbp, int timesteps_to_run) {   
    int ctc = 4;
    int cell_ind = (blockIdx.x * tbp + threadIdx.x) / ctc;
    int cell_type = (blockIdx.x * tbp + threadIdx.x) % ctc;
    // Execute based on cell type
    if (cell_ind >= number_of_cells){return;}
    if (cell_type == 0) {
        th_state_t* state_series = ((th_state_t***)states)[cell_type][cell_ind];
        th_param_t* param = (th_param_t*) params[0];
        for (int timestep = 0; timestep < timesteps_to_run; ++timestep) {
            compute_next_state(state_series + timestep, state_series + timestep + 1, param, dt);
        }
    }
    if (cell_type == 1) {
        stn_state_t* state_series = ((stn_state_t***)states)[cell_type][cell_ind];
        stn_param_t* param = (stn_param_t*) params[0];
        for (int timestep = 0; timestep < timesteps_to_run; ++timestep) {
            compute_next_state(state_series + timestep, state_series + timestep + 1, param, dt);
        }
    }
 }


void BGNetwork::init_states() {
    int cell_counts = this -> sim_params -> cells_per_type;
    this -> states = (void****) malloc(STATE_COUNT * sizeof(void***));
    for (int i = 0; i < STATE_COUNT; ++i) {
        this->states[i] = (void***) malloc(CELL_TYPE_COUNT * sizeof(void***));
        this->states[i][TH] = (void**) malloc(cell_counts * sizeof(void**));
        this->states[i][STN] = (void**) malloc(cell_counts * sizeof(void**));
        hipMallocManaged(&(this->states[i]), CELL_TYPE_COUNT * sizeof(void**));
        hipMallocManaged(&(this->states[i][TH]), cell_counts * sizeof(void*));
        hipMallocManaged(&(this->states[i][STN]), cell_counts * sizeof(void*));
        std::cout << "Assigned and Cuda Malloc Managed STATE " << i << " TH \n";
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            this->states[i][TH][cell_ind] = (th_state_t*) malloc(STEPS_PER_THREAD * sizeof(th_state));
            hipMallocManaged(&(this->states[i][TH][cell_ind]), STEPS_PER_THREAD * sizeof(th_state));
            this->states[i][STN][cell_ind] = (stn_state_t*) malloc(STEPS_PER_THREAD * sizeof(stn_state));
            hipMallocManaged(&(this->states[i][STN][cell_ind]), STEPS_PER_THREAD * sizeof(stn_state));
        }
    }
}

void BGNetwork::init_parameters() {
    this->params = (void**) malloc(CELL_TYPE_COUNT * sizeof(void*)); 
    hipMallocManaged(&(this->params), CELL_TYPE_COUNT * sizeof(void*)); 
    this -> init_th_param();
    this -> init_stn_param();
}

void BGNetwork::init_result_structures() {
    int cell_counts = this->sim_params->cells_per_type;
    double total_dt = this->sim_params->duration / this->sim_params->dt; 
    
    this -> voltage = (double***) malloc(sizeof(double**));
    this -> debug_states = (void***) malloc(sizeof(double**));
    for (int k = 0; k < CELL_TYPE_COUNT; ++k) { 
        (this->voltage)[k]       = (double**) malloc(cell_counts * sizeof(double*));
        (this->debug_states)[k]  = (void**) malloc(cell_counts * sizeof(void*));
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            (this->voltage)[k][cell_ind] = (double*) malloc(total_dt * sizeof(double));
            if (k == TH){(this->debug_states)[k][cell_ind] = (th_state_t*) malloc(total_dt * sizeof(th_state_t));}
            if (k == STN){(this->debug_states)[k][cell_ind] = (stn_state_t*) malloc(total_dt * sizeof(stn_state_t));}
        }
    }
}


void BGNetwork::init_th_param() {
    this->params[TH] = malloc(sizeof(th_param_t));
    hipMallocManaged(&params[TH], sizeof(th_param_t));
    auto th_params = (th_param_t *) params[TH];
    th_params -> C_m = 1.0;
    th_params -> g_L = 0.05;
    th_params -> E_L = -70;
    th_params -> g_Na = 3.0;
    th_params -> E_Na = 50.0;
    th_params -> g_K = 5.0;
    th_params -> E_K = -75.0;
    th_params -> g_T = 5.0;
    th_params -> E_T = 0.0;
    std::cout << "Built Parameter Map" << std::endl;
}


void BGNetwork::init_stn_param() {
    this->params[STN] = malloc(sizeof(stn_param_t));
    hipMallocManaged(&params[STN], sizeof(stn_param_t));
    auto stn_param = (stn_param_t *) params[STN];
    stn_param->C_m = 1.0;
    stn_param->g_L = 2.25;
    stn_param->E_L = -60.0;
    stn_param->g_Na = 37.0;
    stn_param->E_Na = 55.0;
    stn_param->g_K = 45.0;
    stn_param->E_K = -80.0;
    stn_param->g_T = 0.5;
    stn_param->E_T = 0.0;
    stn_param->g_Ca = 2.0;
    stn_param->E_Ca = 140.0;
    stn_param->g_ahp = 20.0;
    stn_param->E_ahp = -80.0;
    std::cout << "Built Parameter Map" << std::endl;
}

void BGNetwork::initialize_cells() {
    auto th_start = (th_state_t**) this->states[0][TH];
    auto stn_start = (stn_state_t**) this->states[0][STN];
    for (int i = 0; i < sim_params-> cells_per_type; ++i) {
        init_state(&th_start[i][0]); //0'th time index of each cell
        init_state(&stn_start[i][0]);
    }
}


BGNetwork::BGNetwork(simulation_parameters_t* sp){
    std::cout << "BGNetwork Constructor" << std::endl;
    this -> sim_params = sp;
    this -> init_states();
    std::cout << "Finished Init States" << std::endl;
    this -> init_parameters();     
    std::cout << "Initialized Parameters" << std::endl;
    this -> initialize_cells();
    std::cout << "Initialized Cells" << std::endl;
    this -> init_result_structures();
    std::cout << "Initialized Result Structures" << std::endl;
}


void BGNetwork::transfer_voltage(void*** from_states, void*** to_states, int from_t, int to_t, int number_of_states) {
    for (int t = 0; t < number_of_states; ++t) {
        for (int cell_ind = 0; cell_ind < this->sim_params->cells_per_type;++cell_ind) {
            ((th_state_t***) to_states)[TH][cell_ind][to_t + t].voltage = ((th_state_t***) from_states) [TH][cell_ind][from_t + t].voltage;
            ((stn_state_t***) to_states)[STN][cell_ind][t].voltage = ((stn_state_t***) from_states) [STN][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPE][cell_ind][t].voltage = ((th_state_t***) from_states) [GPE][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPI][cell_ind][t].voltage = ((th_state_t***) from_states) [GPI][cell_ind][t].voltage;
        }
    }
}


void BGNetwork::transfer_states(void*** from_states, void*** to_states, int from_t, int to_t, int number_of_states) {
    for (int t = 0; t < number_of_states; ++t) {
        for (int cell_ind = 0; cell_ind < this->sim_params->cells_per_type;++cell_ind) {
            ((th_state_t***) to_states)[TH][cell_ind][to_t + t] = ((th_state_t***) from_states) [TH][cell_ind][from_t + t];
            ((stn_state_t***) to_states)[STN][cell_ind][t].voltage = ((stn_state_t***) from_states) [STN][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPE][cell_ind][t].voltage = ((th_state_t***) from_states) [GPE][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPI][cell_ind][t].voltage = ((th_state_t***) from_states) [GPI][cell_ind][t].voltage;
        }
    }
}


void BGNetwork::advance_simulation() {
    /*
    int executions = 0;
    void*** sim_state = this->states[executions  % STATE_COUNT];
    void*** rest_state = this->states[(executions + 1) % STATE_COUNT];
    int block_count = (sim_params->cells_per_type * CELL_TYPE_COUNT) / THREADS_PER_BLOCK;
    if ((sim_params->cells_per_type * CELL_TYPE_COUNT) % THREADS_PER_BLOCK > 0){block_count ++;}
    
    //std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
    advance_step<<<block_count, THREADS_PER_BLOCK>>>(sim_state, params, sim_params->cells_per_type, sim_params->dt, THREADS_PER_BLOCK, STEPS_PER_THREAD - 1);
    hipDeviceSynchronize();
    //std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
   

    //std::cout << "Out of CUDA Call" << std::endl; 
    //std::cout << ((th_state_t**) start_st) << ", " << ((th_state_t**) start_st)[0][0].voltage << std::endl; 
    //std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    
    for (int cell_type = 0; cell_type < CELL_TYPE_COUNT; ++cell_type) {
        for (int cell_ind = 0; cell_ind < sim_params->cells_per_type; ++cell_ind) { // Strangely cell_counts[TH] does not work here?!?! 
            VOLTAGE[cell_type][cell_ind][dt_index] = ((th_state_t**)start_st)[cell_type][cell_ind].voltage;
        }
    }
    dt_index ++;
    void** tmp = start_st;
    start_st = end_st;
    end_st = tmp;
    std::cout << "\r"  << "Iteration: " << dt_index << std::endl;
*/
}
/*
int BGNetwork::debug(th_state_t* state) {
    for (int i = 0; i < sim_params->cells_per_type; ++i) {
        std::ostringstream output_file_name;
        output_file_name << "output/TH_NEURON_" << i << ".txt";
        std::ofstream out(output_file_name.str(), std::ios::app);
        out << "DT=" << dt << ", " << get_debug_string(&state[i]) << std::endl;
        out.close();
    }
    return 0;
}
*/

int BGNetwork::simulate() {
    return 0;
}

int BGNetwork::simulate_debug() {
    int total_steps = this->sim_params->duration / this->sim_params->dt;
    int step = 0;
    int cycles= 0;
    std::cout << "Total Steps: " << total_steps << std::endl;
    while(step < total_steps) {
        int cycle_steps = total_steps - step;
        if (cycle_steps >= STEPS_PER_THREAD-1) {cycle_steps = STEPS_PER_THREAD-1;}
     
        void*** sim_state = this->states[cycles  % STATE_COUNT];
        void*** rest_state = this->states[(cycles - 1) % STATE_COUNT];
        int block_count = (sim_params->cells_per_type * CELL_TYPE_COUNT) / THREADS_PER_BLOCK;
        if ((sim_params->cells_per_type * CELL_TYPE_COUNT) % THREADS_PER_BLOCK > 0){block_count ++;}
        
        std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
        advance_step<<<block_count, THREADS_PER_BLOCK>>>(sim_state, params, sim_params->cells_per_type, sim_params->dt, THREADS_PER_BLOCK, cycle_steps);
        if (cycles > 0) {this -> transfer_states(this->debug_states, this->debug_states, step, step, cycle_steps);}        
        hipDeviceSynchronize();
        std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
        cycles++;
        step += cycle_steps;
        if (step == total_steps) {this -> transfer_states(sim_state, this->debug_states, 0, step, cycle_steps);}
    }
    
    std::cout<< "End of Simulate Debug" << std::endl;
    return 0;
}


double*** BGNetwork::get_voltage() {
    return this->voltage;
}


void*** BGNetwork::get_debug_states() {
    return this -> debug_states;
}
