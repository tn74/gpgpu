#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-04-04.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include "BGNetwork.h"
#include "THNeuron.h"
#include "STNNeuron.h"

__global__ 
 void advance_step(void*** states, void** params, int number_of_cells,  double dt, int tbp, int timesteps_to_run) {   
    int ctc = 4;
    int cell_ind = (blockIdx.x * tbp + threadIdx.x) / ctc;
    int cell_type = (blockIdx.x * tbp + threadIdx.x) % ctc;
    // Execute based on cell type
    if (cell_ind >= number_of_cells){return;}
    if (cell_type == 0) {
        th_state_t* state_series = ((th_state_t***)states)[cell_type][cell_ind];
        th_param_t* param = (th_param_t*) params[0];
        for (int timestep = 0; timestep < timesteps_to_run; ++timestep) {
            compute_next_state(state_series + timestep, state_series + timestep + 1, param, dt);
        }
    }
    if (cell_type == 1) {
        stn_state_t* state_series = ((stn_state_t***)states)[cell_type][cell_ind];
        stn_param_t* param = (stn_param_t*) params[1];
        for (int timestep = 0; timestep < timesteps_to_run; ++timestep) {
            compute_next_state(state_series + timestep, state_series + timestep + 1, param, dt);
        }
    }
 }


void BGNetwork::init_states() {
    int cell_counts = this -> sim_params -> cells_per_type;
    this -> states = (void****) malloc(STATE_COUNT * sizeof(void***));
    for (int i = 0; i < STATE_COUNT; ++i) {
        this->states[i] = (void***) malloc(CELL_TYPE_COUNT * sizeof(void***));
        this->states[i][TH] = (void**) malloc(cell_counts * sizeof(void**));
        this->states[i][STN] = (void**) malloc(cell_counts * sizeof(void**));
        hipMallocManaged(&(this->states[i]), CELL_TYPE_COUNT * sizeof(void**));
        hipMallocManaged(&(this->states[i][TH]), cell_counts * sizeof(void*));
        hipMallocManaged(&(this->states[i][STN]), cell_counts * sizeof(void*));
        std::cout << "Assigned and Cuda Malloc Managed STATE " << i << " TH \n";
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            this->states[i][TH][cell_ind] = (th_state_t*) malloc(STEPS_PER_THREAD * sizeof(th_state));
            hipMallocManaged(&(this->states[i][TH][cell_ind]), STEPS_PER_THREAD * sizeof(th_state));
            this->states[i][STN][cell_ind] = (stn_state_t*) malloc(STEPS_PER_THREAD * sizeof(stn_state));
            hipMallocManaged(&(this->states[i][STN][cell_ind]), STEPS_PER_THREAD * sizeof(stn_state));
        }
    }
}

void BGNetwork::init_parameters() {
    this->params = (void**) malloc(CELL_TYPE_COUNT * sizeof(void*)); 
    hipMallocManaged(&(this->params), CELL_TYPE_COUNT * sizeof(void*)); 
    
    this->params[TH] = malloc(sizeof(th_param_t));
    hipMallocManaged(&params[TH], sizeof(th_param_t));
    auto th_param = (th_param_t *) params[TH];
    init_th_param(th_param);
    
    this->params[STN] = malloc(sizeof(stn_param_t));
    hipMallocManaged(&params[STN], sizeof(stn_param_t));
    auto stn_param = (stn_param_t *) params[STN];
    init_stn_param(stn_param);
}

void BGNetwork::init_result_structures() {
    int cell_counts = this->sim_params->cells_per_type;
    double total_dt = this->sim_params->duration / this->sim_params->dt; 
    
    this -> voltage = (double***) malloc(CELL_TYPE_COUNT * sizeof(double**));
    this -> debug_states = (void***) malloc(CELL_TYPE_COUNT * sizeof(void**));
    for (int k = 0; k < CELL_TYPE_COUNT; ++k) { 
        (this->voltage)[k]       = (double**) malloc(cell_counts * sizeof(double*));
        (this->debug_states)[k]  = (void**) malloc(cell_counts * sizeof(void*));
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            (this->voltage)[k][cell_ind] = (double*) malloc(total_dt * sizeof(double));
            if (k == TH){(this->debug_states)[k][cell_ind] = (th_state_t*) malloc(total_dt * sizeof(th_state_t));}
            if (k == STN){(this->debug_states)[k][cell_ind] = (stn_state_t*) malloc(total_dt * sizeof(stn_state_t));}
        }
    }
}



void BGNetwork::initialize_cells() {
    auto th_start = (th_state_t**) this->states[0][TH];
    auto stn_start = (stn_state_t**) this->states[0][STN];
    for (int i = 0; i < sim_params-> cells_per_type; ++i) {
        init_state(&th_start[i][0]); //0'th time index of each cell
        init_state(&stn_start[i][0]);
    }
}


BGNetwork::BGNetwork(simulation_parameters_t* sp){
    std::cout << "BGNetwork Constructor" << std::endl;
    this -> sim_params = sp;
    this -> init_states();
    std::cout << "Finished Init States" << std::endl;
    this -> init_parameters();     
    std::cout << "Initialized Parameters" << std::endl;
    this -> initialize_cells();
    std::cout << "Initialized Cells" << std::endl;
    this -> init_result_structures();
    std::cout << "Initialized Result Structures" << std::endl;
}


void BGNetwork::transfer_voltage(void*** from_states, void*** to_states, int from_t, int to_t, int number_of_states) {
    for (int t = 0; t < number_of_states; ++t) {
        for (int cell_ind = 0; cell_ind < this->sim_params->cells_per_type;++cell_ind) {
            ((th_state_t***) to_states)[TH][cell_ind][to_t + t].voltage = ((th_state_t***) from_states) [TH][cell_ind][from_t + t].voltage;
            ((stn_state_t***) to_states)[STN][cell_ind][t].voltage = ((stn_state_t***) from_states) [STN][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPE][cell_ind][t].voltage = ((th_state_t***) from_states) [GPE][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPI][cell_ind][t].voltage = ((th_state_t***) from_states) [GPI][cell_ind][t].voltage;
        }
    }
}


void BGNetwork::transfer_states(void*** from_states, void*** to_states, int from_t, int to_t, int number_of_states) {
    for (int t = 0; t < number_of_states; ++t) {
        for (int cell_ind = 0; cell_ind < this->sim_params->cells_per_type;++cell_ind) {
            ((th_state_t***) to_states)[TH][cell_ind][to_t + t] = ((th_state_t***) from_states) [TH][cell_ind][from_t + t];
            ((stn_state_t***) to_states)[STN][cell_ind][t].voltage = ((stn_state_t***) from_states) [STN][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPE][cell_ind][t].voltage = ((th_state_t***) from_states) [GPE][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPI][cell_ind][t].voltage = ((th_state_t***) from_states) [GPI][cell_ind][t].voltage;
        }
    }
}

int BGNetwork::simulate() {
    return 0;
}

int BGNetwork::simulate_debug() {
    int total_steps = this->sim_params->duration / this->sim_params->dt;
    int step = 0;
    int cycles= 0;
    std::cout << "Total Steps: " << total_steps << std::endl;
    while(step < total_steps) {
        int cycle_steps = total_steps - step;
        if (cycle_steps >= STEPS_PER_THREAD-1) {cycle_steps = STEPS_PER_THREAD-1;}
     
        void*** sim_state = this->states[cycles  % STATE_COUNT];
        void*** rest_state = this->states[(cycles + 1) % STATE_COUNT];
        int block_count = (sim_params->cells_per_type * CELL_TYPE_COUNT) / THREADS_PER_BLOCK;
        if ((sim_params->cells_per_type * CELL_TYPE_COUNT) % THREADS_PER_BLOCK > 0){block_count ++;}
        
        std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
        advance_step<<<block_count, THREADS_PER_BLOCK>>>(sim_state, params, sim_params->cells_per_type, sim_params->dt, THREADS_PER_BLOCK, cycle_steps);
        hipDeviceSynchronize();
        this -> transfer_states(sim_state, this->debug_states, 0, step, cycle_steps);        
        std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
        this -> transfer_states(sim_state, rest_state, cycle_steps, 0, 1);
        
        cycles++;
        step += cycle_steps;
        if (step == total_steps) {this -> transfer_states(sim_state, this->debug_states, 0, step, cycle_steps);}
    }
    
    std::cout<< "End of Simulate Debug" << std::endl;
    return 0;
}


double*** BGNetwork::get_voltage() {
    return this->voltage;
}


void*** BGNetwork::get_debug_states() {
    return this -> debug_states;
}
