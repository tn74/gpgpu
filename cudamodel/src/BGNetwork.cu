#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-04-04.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include "BGNetwork.h"
#include "THNeuron.h"
#include <chrono>
#include <thread>
__global__ 
 void advance_step(void*** states, void** params, int number_of_cells,  double dt, int tbp, int timesteps_to_run) {   
    int ctc = 4;
    int cell_ind = (blockIdx.x * tbp + threadIdx.x) / ctc;
    int cell_type = (blockIdx.x * tbp + threadIdx.x) % ctc;
    // Execute based on cell type
    if (cell_ind >= number_of_cells){return;}
    if (cell_type == 0) {
        th_state_t* state_series = ((th_state_t***)states)[cell_type][cell_ind];
        th_param_t* param = (th_param_t*) params[0];
        for (int timestep = 0; timestep < timesteps_to_run; ++timestep) {
            compute_next_state(state_series + timestep, state_series + timestep + 1, param, dt);
        }
    }
 }


void BGNetwork::init_states() {
    int cell_counts = this -> sim_params -> cells_per_type;
    this -> states = (void****) malloc(STATE_COUNT * sizeof(void***));
    for (int i = 0; i < STATE_COUNT; ++i) {
        this->states[i] = (void***) malloc(CELL_TYPE_COUNT * sizeof(void***));
        this->states[i][TH] = (void**) malloc(cell_counts * sizeof(void**));
        hipMallocManaged(&(this->states[i]), CELL_TYPE_COUNT * sizeof(void**));
        hipMallocManaged(&(this->states[i][TH]), cell_counts * sizeof(void*));
        std::cout << "Assigned and Cuda Malloc Managed STATE " << i << " TH \n";
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            std::cout << "Size of th_state_t " << sizeof(th_state_t) << " bytes" << "Size of th_state " << sizeof(th_state) << " bytes\n";    
            this->states[i][TH][cell_ind] = (th_state_t*) malloc(STEPS_PER_THREAD * sizeof(th_state));
            hipMallocManaged(&(this->states[i][TH][cell_ind]), STEPS_PER_THREAD * sizeof(th_state));
        }
    }
}

void BGNetwork::init_parameters() {
    this->params = (void**) malloc(CELL_TYPE_COUNT * sizeof(void*)); 
    hipMallocManaged(&(this->params), CELL_TYPE_COUNT * sizeof(void*)); 
    this -> init_th_param();
}

void BGNetwork::init_result_structures() {
    int cell_counts = this->sim_params->cells_per_type;
    double total_dt = this->sim_params->duration / this->sim_params->dt; 
    
    this -> voltage = (double***) malloc(sizeof(double**));
    this -> debug_states = (void***) malloc(sizeof(double**));
    for (int k = 0; k < CELL_TYPE_COUNT; ++k) { 
        (this->voltage)[k]       = (double**) malloc(cell_counts * sizeof(double*));
        (this->debug_states)[k]  = (void**) malloc(cell_counts * sizeof(void*));
        for (int cell_ind = 0; cell_ind < cell_counts; ++cell_ind) {
            (this->voltage)[k][cell_ind] = (double*) malloc(total_dt * sizeof(double));
            if (k == TH){(this->debug_states)[k][cell_ind] = (th_state_t*) malloc(total_dt * sizeof(th_state_t));}
        }
    }
}


void BGNetwork::init_th_param() {
    this->params[TH] = malloc(sizeof(th_param_t));
    hipMallocManaged(&params[TH], sizeof(th_param_t));
    auto th_params = (th_param_t *) params[TH];
    th_params -> C_m = 1.0;
    th_params -> g_L = 0.05;
    th_params -> E_L = -70;
    th_params -> g_Na = 3.0;
    th_params -> E_Na = 50.0;
    th_params -> g_K = 5.0;
    th_params -> E_K = -75.0;
    th_params -> g_T = 5.0;
    th_params -> E_T = 0.0;
    std::cout << "Built Parameter Map" << std::endl;
}


void BGNetwork::initialize_cells() {
    auto th_start = (th_state_t**) this->states[0][TH];
    for (int i = 0; i < sim_params-> cells_per_type; ++i) {
        init_state(&th_start[i][0]); //0'th time index of each cell
    }
}


BGNetwork::BGNetwork(simulation_parameters_t* sp){
    std::cout << "BGNetwork Constructor" << std::endl;
    this -> sim_params = sp;
    this -> init_states();
    std::cout << "Finished Init States" << std::endl;
    this -> init_parameters();     
    std::cout << "Initialized Parameters" << std::endl;
    this -> initialize_cells();
    std::cout << "Initialized Cells" << std::endl;
    this -> init_result_structures();
    std::cout << "Initialized Result Structures" << std::endl;
}


void BGNetwork::transfer_voltage(void*** from_states, void*** to_states, int from_t, int to_t, int number_of_states) {
    for (int t = 0; t < number_of_states; ++t) {
        for (int cell_ind = 0; cell_ind < this->sim_params->cells_per_type;++cell_ind) {
            ((th_state_t***) to_states)[TH][cell_ind][to_t + t].voltage = ((th_state_t***) from_states) [TH][cell_ind][from_t + t].voltage;
            //((th_state_t***) to_states)[STN][cell_ind][t].voltage = ((th_state_t***) from_states) [STN][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPE][cell_ind][t].voltage = ((th_state_t***) from_states) [GPE][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPI][cell_ind][t].voltage = ((th_state_t***) from_states) [GPI][cell_ind][t].voltage;
        }
    }
}


void BGNetwork::transfer_states(void*** from_states, void*** to_states, int from_t, int to_t, int number_of_states) {
    for (int t = 0; t < number_of_states; ++t) {
        for (int cell_ind = 0; cell_ind < this->sim_params->cells_per_type;++cell_ind) {
            ((th_state_t***) to_states)[TH][cell_ind][to_t + t] = ((th_state_t***) from_states) [TH][cell_ind][from_t + t];
            //((th_state_t***) to_states)[STN][cell_ind][t].voltage = ((th_state_t***) from_states) [STN][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPE][cell_ind][t].voltage = ((th_state_t***) from_states) [GPE][cell_ind][t].voltage;
            //((th_state_t***) to_states)[GPI][cell_ind][t].voltage = ((th_state_t***) from_states) [GPI][cell_ind][t].voltage;
        }
    }
}

int BGNetwork::simulate() {
    return 0;
}

int BGNetwork::simulate_debug() {
    int total_steps = this->sim_params->duration / this->sim_params->dt;
    int step = 0;
    int cycles= 0;
    std::cout << "Total Steps: " << total_steps << std::endl;
    while(step < total_steps) {
        int cycle_steps = total_steps - step;
        if (cycle_steps >= STEPS_PER_THREAD-1) {cycle_steps = STEPS_PER_THREAD-1;}
     
        void*** sim_state = this->states[cycles  % STATE_COUNT];
        void*** rest_state = this->states[(cycles + 1) % STATE_COUNT];
        int block_count = (sim_params->cells_per_type * CELL_TYPE_COUNT) / THREADS_PER_BLOCK;
        if ((sim_params->cells_per_type * CELL_TYPE_COUNT) % THREADS_PER_BLOCK > 0){block_count ++;}
        
        std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
        advance_step<<<block_count, THREADS_PER_BLOCK>>>(sim_state, params, sim_params->cells_per_type, sim_params->dt, THREADS_PER_BLOCK, cycle_steps);
        hipDeviceSynchronize();
        this -> transfer_states(sim_state, this->debug_states, 0, step, cycle_steps);        
        std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError()) << std::endl;
        this -> transfer_states(sim_state, rest_state, cycle_steps, 0, 1);
        
        cycles++;
        step += cycle_steps;
        if (step == total_steps) {this -> transfer_states(sim_state, this->debug_states, 0, step, cycle_steps);}
    }
    
    std::cout<< "End of Simulate Debug" << std::endl;
    return 0;
}


double*** BGNetwork::get_voltage() {
    return this->voltage;
}


void*** BGNetwork::get_debug_states() {
    return this -> debug_states;
}
