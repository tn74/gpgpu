#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-04-04.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include "BGNetwork.h"
#include "THNeuron.h"
#include <chrono>
#include <thread>
__global__ 
 void advance_step(void** start_state, void** end_state, void** params, int* cell_counts, double dt, int tbp) {   
     // Determine what start to end this is supposed to compute
    int cell_ind = blockIdx.y * tbp + threadIdx.x;
    int cell_type = blockIdx.x;
    if (cell_ind >= cell_counts[cell_type]) {return;}
    
    // Execute based on cell type
//    if (cell_type == TH) {
        th_state_t* start = &((th_state_t**)start_state)[TH][cell_ind];
        th_state_t* end = &((th_state_t**) end_state)[TH][cell_ind];
        th_param_t* param = ((th_param_t**) params)[TH];
//    }
    compute_next_state(start, end, param, dt);
}


BGNetwork::BGNetwork(simulation_parameters_t* sp){
    std::cout << "BGNetwork Constructor" << std::endl;
    sim_params = sp;
    dt_index = 0;
    size_t CTC_SIZE = CELL_TYPE_COUNT * sizeof(void*);
    params = (void**) malloc(CTC_SIZE); hipMallocManaged(&params, CTC_SIZE);
    start_st = (void**) malloc(CTC_SIZE); hipMallocManaged(&start_st, CTC_SIZE);
    end_st  = (void**) malloc(CTC_SIZE); hipMallocManaged(&end_st, CTC_SIZE);
    cell_counts  = (int*) malloc(CELL_TYPE_COUNT * sizeof(int*)); hipMallocManaged(&cell_counts, CELL_TYPE_COUNT * sizeof(int*));
    for (int c = 0; c < CELL_TYPE_COUNT; ++c) {cell_counts[c] = sp->cells_per_type;}
    std::cout << cell_counts[TH] << std::endl;
    start_st[TH] = malloc(cell_counts[TH] * sizeof(th_state_t)); hipMallocManaged(&start_st[TH], cell_counts[TH] * sizeof(th_state_t));
    end_st[TH] = malloc(cell_counts[TH]  * sizeof(th_state_t)); hipMallocManaged(&end_st[TH], cell_counts[TH] * sizeof(th_state_t));
    hipMallocManaged(&start_st[0], sizeof(th_state_t));

    VOLTAGE = (double***) malloc(CTC_SIZE);
    std::cout << "Voltage[TH]: " << VOLTAGE[TH] << std::endl;
    double total_dt = sp->duration / sp->dt; 
    for (int k = 0; k < CELL_TYPE_COUNT; ++k) { 
        VOLTAGE[k] = (double**) malloc(cell_counts[k] * sizeof(double*));
            for (int i = 0; i < cell_counts[k]; ++i) {
                VOLTAGE[k][i] = (double*) malloc(total_dt * sizeof(double));
            } 
    }
    build_parameter_map();
    initialize_cells();

}

void BGNetwork::build_parameter_map() {
    params[TH] = malloc(sizeof(th_param));
    hipMallocManaged(&params[TH], sizeof(th_param));
    auto th_params = (th_param_t *) params[TH];
    th_params -> C_m = 1.0;
    th_params -> g_L = 0.05;
    th_params -> E_L = -70;
    th_params -> g_Na = 3.0;
    th_params -> E_Na = 50.0;
    th_params -> g_K = 5.0;
    th_params -> E_K = -75.0;
    th_params -> g_T = 5.0;
    th_params -> E_T = 0.0;
    std::cout << "Built Parameter Map" << std::endl;
}

void BGNetwork::initialize_cells() {
    auto th_start = (th_state_t*) start_st[TH];
    for (int i = 0; i < sim_params-> cells_per_type; ++i) {
        init_state(&th_start[i]);
    }
}

void BGNetwork::advance_time_step() {
    dim3 grid(CELL_TYPE_COUNT, sim_params -> cells_per_type / THREADS_PER_BLOCK + 1);
    advance_step<<<grid, THREADS_PER_BLOCK>>>(start_st, end_st, params, cell_counts, sim_params->dt, THREADS_PER_BLOCK);
    hipDeviceSynchronize();
    std::cout << "Out of CUDA Call" << std::endl; 
    std::cout << ((th_state_t**) start_st) << ", " << ((th_state_t**) start_st)[0][0].voltage << std::endl; 
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    for (int cell_type = 0; cell_type < CELL_TYPE_COUNT; ++cell_type) {
        for (int cell_ind = 0; cell_ind < sim_params->cells_per_type; ++cell_ind) { // Strangely cell_counts[TH] does not work here?!?! 
            VOLTAGE[cell_type][cell_ind][dt_index] = ((th_state_t**)start_st)[cell_type][cell_ind].voltage;
        }
    }
    dt_index ++;
    void** tmp = start_st;
    start_st = end_st;
    end_st = tmp;
    std::cout << "\r"  << "Iteration: " << dt_index << std::endl;
}

void BGNetwork::debug(th_state_t* state) {
    for (int i = 0; i < sim_params->cells_per_type; ++i) {
        std::ostringstream output_file_name;
        output_file_name << "output/TH_NEURON_" << i << ".txt";
        std::ofstream out(output_file_name.str(), std::ios::app);
        out << "DT=" << dt_index << ", " << get_debug_string(&state[i]) << std::endl;
        out.close();
    }
}

int BGNetwork::simulate() {
    for (int i = 0; i < sim_params->duration / sim_params->dt; ++i) {
        advance_time_step();
    }
    std::cout<< "End of Simulate" << std::endl;
    return 0;
};

double*** BGNetwork::get_voltage() {
    return VOLTAGE;
}
