#include "hip/hip_runtime.h"
//
// Created by Jasmine Lu on 2019-04-27.
//

#include "STNNeuron.h"
#include "gating.h"


__device__ void compute_next_state(stn_state_t *in, stn_state_t *out, stn_param_t *params, double dt){
    compute_currents(in, out, params);
    compute_gating(in, out, params, dt);
    
    out -> voltage = params -> g_L;
    double current_sum =
            out -> I_L +
            out -> I_Na +
            out -> I_K +
            out -> I_T +
            out -> I_Ca +
            out -> I_ahp;
    out -> voltage = in -> voltage + dt * current_sum / params->C_m;
}

__device__ void compute_currents(stn_state_t *in, stn_state_t *out, stn_param_t *params){
    out->I_L = -(params->g_L * (in->voltage - params->E_L));
    out->I_Na = -(params->g_Na * (pow(stn_minf(in->voltage), 3)) * in->H * (in->voltage- params->E_Na));
    out->I_K = -(params->g_K * pow(in->N, 4) * (in->voltage- params->E_K));
    out->I_T = -(params->g_T * pow(stn_ainf(in->voltage), 3) * pow(stn_binf(in->R), 2) * in->R * (in->voltage- params->E_Ca));
    out->I_Ca = -(params->g_Ca * pow(in->C, 2) * (in->voltage- params->E_Ca));
    out->I_ahp = -(params->g_ahp * (in->voltage- params->E_ahp) * (in->CA/(in->CA + 15)));
}
__device__ void compute_gating(stn_state_t *in, stn_state_t *out, stn_param_t *params, double dt){
    out->H = in->H + dt * 0.75 * ((stn_hinf(in->voltage) - in->H)/stn_tauh(in->voltage));
    out->R = in->R + dt * 0.2 * (stn_rinf(in->voltage) - in->R)/stn_taur(in->voltage);
    out->N = in->N + dt * 0.75 * (stn_ninf(in->voltage) - in->N)/stn_taun(in->voltage);
    out->C = in->C + dt * 0.08 * (stn_cinf(in->voltage) - in->C)/stn_tauc(in->voltage);
    out->CA = in->CA + dt * 3.75 * pow(10, -5) * (-(in->I_Ca) - in->I_T - (22.5 * in->CA));
}
void init_state(stn_state_t *in){
    in->voltage = -57.0;
    in->H = stn_hinf(in->voltage);
    in->R = stn_rinf(in->voltage);
    in->N = stn_ninf(in->voltage);
    in->C = stn_cinf(in->voltage);
    in->CA = 0.1;
}

void init_stn_param(stn_param_t* param) {
    param->C_m = 1.0;
    param->g_L = 2.25;
    param->E_L = -60.0;
    param->g_Na = 37.0;
    param->E_Na = 55.0;
    param->g_K = 45.0;
    param->E_K = -80.0;
    param->g_T = 0.5;
    param->E_T = 0.0;
    param->g_Ca = 2.0;
    param->E_Ca = 140.0;
    param->g_ahp = 20.0;
    param->E_ahp = -80.0;
}
