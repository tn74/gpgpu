#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-03-28.
//


#include <string>
#include <iostream>
#include <string>
#include <sstream>
#include <iomanip>
#include <math.h>
#include "THNeuron.cuh"
#include "gating.cuh"

__global__ void trap() {
    int x = 1;
}
 
void compute_next_state(th_state_t *in, th_state_t *out, th_param_t *params, double dt) {
    trap<<<1, 1>>>();
    compute_currents(in, out, params);
    compute_gating(in, out, params, dt);
    double current_sum =
            out -> I_L +
            out -> I_Na +
            out -> I_K +
            out -> I_T;
    out -> voltage = in -> voltage + dt * current_sum / params->C_m;

}
void compute_currents(th_state_t *in, th_state_t *out, th_param_t *p){
    double v = in->voltage;
    out->I_K = -(p->g_K * pow(0.75 *(1 - in->H), 4) * (v - p->E_K));
    out->I_L = -1 * p->g_L * (v - p->E_L);
    out->I_Na = -1 * p->g_Na * pow(th_minf(v), 3) * in->H * (v - p->E_Na);
    out->I_T = -p->g_T * pow(th_pinf(v), 2) * in->R * (v - p->E_T);

}

void compute_gating(th_state_t *in, th_state_t *out, th_param_t *params, double dt){
    double v = in->voltage;
    out->H = in->H + dt * (th_hinf(v) - in->H)/th_tauh(v);
    out->R = in->R + dt * (th_rinf(v) - in->R)/th_taur(v);
}

void init_state(th_state_t *in){
    in->voltage = -57.0;
    in->H = th_hinf(in->voltage);
    in->R = th_rinf(in->voltage);
    in -> I_K = 0;
    in -> I_L = 0;
    in -> I_Na = 0;
    in -> I_T = 0;
}

std::string get_debug_string(th_state_t* state) {
    std::ostringstream debug_str;
    debug_str.setf(std::ios::fixed, std::ios::floatfield);
    debug_str << std::setprecision(15);
    debug_str
    << "VOLTAGE=" << state->voltage
    << ", I_K=" << state -> I_K
    << ", I_L=" << state -> I_L
    << ", I_Na=" << state -> I_Na
    << ", I_T=" << state -> I_T
    << ", H=" << state -> H
    << ", R=" << state -> R;
    return debug_str.str();
}
