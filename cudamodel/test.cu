#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include "src/THNeuron.h"
#include "src/BGNetwork.h"
#include "dbs.cu"


extern "C" double*** test_run() { 
    auto sim_params = (simulation_parameters_t *) malloc(sizeof(simulation_parameters));
    sim_params->cells_per_type = 10;
    sim_params->duration = 1;
    sim_params->dt = 0.01;
    return execute_simulation(sim_params);
}

__global__
void compute_preceeding(int* arr, int* sumarr, int maxlen) {
    int maxind = 64 * blockIdx.x + threadIdx.x;
    if (maxind >= maxlen) {
        return;
    }
    sumarr[maxind] = 0;
    for (int i = 0; i < maxind; ++i) {
        sumarr[maxind] += arr[i];
    } 
}

extern "C" int cudatest() {
    int arrlen = 90000;
    int* arr = (int*) malloc(sizeof(int) * arrlen);
    int* sumarr = (int*) malloc(sizeof(int) * arrlen); 
    hipMallocManaged(&arr, arrlen*sizeof(int));
    hipMallocManaged(&sumarr, arrlen*sizeof(int));
    for (int i = 0; i < arrlen; ++i) {
        arr[i] = i;   
        sumarr[i] = 0;
    }
    for (int i = 0; i < arrlen; ++i) {
        std::cout << sumarr[i] << " ";    
    }
    std::cout << std::endl;
    std::cout << "Past Init" << std::endl;
    //compute_preceeding<<<arrlen/64 + 1, 64>>>(arr, sumarr, arrlen);
    
    for (int tix = 0; tix < arrlen; ++tix) {
        for (int i = 0; i < tix; ++i) {
            sumarr[tix] += arr[i];
        }
    }
    
    std::cout << "Past Trap" << std::endl;
    hipDeviceSynchronize();
    for (int i = 0; i < arrlen; ++i) {
        std::cout << sumarr[i] << " ";    
    }
    std::cout << std::endl;
    return 0;
}

extern "C" int plain() {
    return 1;
}

extern "C" int arrsum(int* arr, int arrlen) {
    int s = 0;
    for (int i = 0; i < arrlen; ++i) {
        s += arr[i];
    }
    return s;
}


