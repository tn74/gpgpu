#include "hip/hip_runtime.h"
//
// Created by Trishul Nagenalli on 2019-11-21.
//

#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include "src/THNeuron.h"
#include "src/BGNetwork.h"


extern "C"  int execute_simulation(simulation_parameters_t* sim_params) {
    auto net = BGNetwork(sim_params);
    net.simulate();
    return 0;
}

/*
__global__
void compute_preceeding(int* arr, int* sumarr, int maxlen) {
    int maxind = 64 * blockIdx.x + threadIdx.x;
    if (maxind >= maxlen) {
        return;
    }
    sumarr[maxind] = 0;
    for (int i = 0; i < maxind; ++i) {
        sumarr[maxind] += arr[i];
    } 
}

extern "C" int cudatest() {
    int arrlen = 90000;
    int* arr = (int*) malloc(sizeof(int) * arrlen);
    int* sumarr = (int*) malloc(sizeof(int) * arrlen); 
    hipMallocManaged(&arr, arrlen*sizeof(int));
    hipMallocManaged(&sumarr, arrlen*sizeof(int));
    for (int i = 0; i < arrlen; ++i) {
        arr[i] = i;   
        sumarr[i] = 0;
    }
    for (int i = 0; i < arrlen; ++i) {
        std::cout << sumarr[i] << " ";    
    }
    std::cout << std::endl;
    std::cout << "Past Init" << std::endl;
    //compute_preceeding<<<arrlen/64 + 1, 64>>>(arr, sumarr, arrlen);
    
    for (int tix = 0; tix < arrlen; ++tix) {
        for (int i = 0; i < tix; ++i) {
            sumarr[tix] += arr[i];
        }
    }
    
    std::cout << "Past Trap" << std::endl;
    hipDeviceSynchronize();
    for (int i = 0; i < arrlen; ++i) {
        std::cout << sumarr[i] << " ";    
    }
    std::cout << std::endl;
    return 0;
}

extern "C" int plain() {
    return 1;
}

extern "C" int arrsum(int* arr, int arrlen) {
    int s = 0;
    for (int i = 0; i < arrlen; ++i) {
        s += arr[i];
    }
    return s;
}

extern "C" int addnum(int a, int b) {
    return a + b;
}

extern "C" int plain2() {
    return 2;
}

extern "C"  int execute_simulation(simulation_parameters_t* sim_params) {
    auto net = BGNetwork(sim_params);
    net.simulate();
    return 0;
}
*/
