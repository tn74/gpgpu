#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "gating.cuh"

double gpe_ainf(double V)
{
  return 1/(1+exp(-(V+57)/2));
}

double gpe_hinf(double V)
{
  return 1/(1+exp((V+58)/12));
}

double gpe_minf(double V)
{
  return 1/(1+exp(-(V+37)/10));
}

double gpe_ninf(double V)
{
  return 1/(1+exp(-(V+50)/14));
}

double gpe_rinf(double V)
{
  return 1/(1+exp((V+70)/2));
}

double gpe_sinf(double V)
{
  return 1/(1+exp(-(V+35)/2));
}

double gpe_tauh(double V)
{
  return 0.05+0.27/(1+exp(-(V+40)/-12));
}

double gpe_taun(double V)
{
  return 0.05+0.27/(1+exp(-(V+40)/-12));
}

double Hinf(double V)
{
  return 1/(1+exp(-(V+57)/2));
}

double stn_ainf(double V)
{
  return 1/(1+exp(-(V+63)/7.8));
}

double stn_binf(double R)
{
  return 1/(1+exp(-(R-0.4)/0.1))-1/(1+exp(0.4/0.1));
}

double stn_cinf(double V)
{
  return 1/(1+exp(-(V+20)/8));
}

double stn_hinf(double V)
{
  return 1/(1+exp((V+39)/3.1));
}

double stn_minf(double V)
{
  return 1/(1+exp(-(V+30)/15));
}

double stn_ninf(double V)
{
  return 1/(1+exp(-(V+32)/8.0));
}

double stn_rinf(double V)
{
  return 1/(1+exp((V+67)/2));
}

double stn_sinf(double V)
{
  return 1/(1+exp(-(V+39)/8));
}

double stn_tauc(double V)
{
  return 1 + 10/(1+exp(-(V+80)/26));
}

double stn_tauh(double V)
{
  return 1 + 500/(1+exp(-(V+57)/-3));
}

double stn_taun(double V)
{
  return 1 + 100/(1+exp(-(V+80)/-26));
}

double stn_taur(double V)
{
  return 7.1 + 17.5/(1+exp(-(V-68)/-2.2));
}

double th_hinf(double V)
{
  return 1/(1+exp((V+41)/4));
}

double th_minf(double V)
{
  return 1/(1+exp(-(V+37)/7));
}

double th_pinf(double V)
{
  return 1/(1+exp(-(V+60)/6.2));
}

double th_rinf(double V)
{
  return 1/(1+exp((V+84)/4));
}

double ah(double V)
{
  return 0.128*exp(-(V+46)/18);
}

double bh(double V)
{
  return 4/(1+exp(-(V+23)/5));
}

double th_tauh(double V)
{
  return 1/(ah(V)+bh(V));
}

double th_taur(double V)
{
  return 0.15*(28+exp(-(V+25)/10.5));
}
