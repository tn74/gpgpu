#include "hip/hip_runtime.h"
#include <iostream>
#include "src/parameter_structs.h"

struct TH_PARAM {
    double c_m;
    TH_PARAM() {
        c_m=109;
    }
};

__global__ void func(int* num_array) {
    num_array[threadIdx.x] = threadIdx.x;
    return;
}

void struct_test() {
    th_param_t* s;
    hipMallocManaged(&s, sizeof(th_param));
    initialize_th_param(s);
    std::cout << s->c_m << std::endl;
}


int main() {
    struct_test();
    int* num_ptr = (int*) malloc(255 * sizeof(int));
    std::cout << *num_ptr << std::endl;
//    func <<<1,  32>>> (num_ptr);
    std::cout << *num_ptr << std::endl;
    return 0;
}