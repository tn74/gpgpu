
#include <hip/hip_runtime.h>
#include <iostream>

struct TH_PARAM {
    double c_m;
    TH_PARAM() {
        c_m=109;
    }
};

__global__ void func(int* num_array) {
    num_array[threadIdx.x] = threadIdx.x;
    return;
}


int main() {
    int* num_ptr = (int*) malloc(255 * sizeof(int));
    std::cout << *num_ptr << std::endl;
    func <<<1,  32>>> (num_ptr);
    std::cout << *num_ptr << std::endl;
    return 0;
}