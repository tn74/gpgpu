
#include <hip/hip_runtime.h>
#include <iostream>

struct TH_PARAM {
    double c_m;
    TH_PARAM() {
        c_m=109;
    }
};

__global__ void func(int* num_ptr) {
    int &num = *num_ptr;
    num = num + 1;
    return;
}


int main() {
    int* num_ptr = (int*) malloc(sizeof(int));
    (*num_ptr) = 10;
    std::cout << *num_ptr << std::endl;
    func <<<1,  32>>> (num_ptr);
    std::cout << *num_ptr << std::endl;
    return 0;
}